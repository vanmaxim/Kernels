#include <hip/hip_runtime.h>
#include <stdio.h>

int const TILE_WIDTH = 2;

__global__
void TiledMatrixMulKernel(float *M, float *N, float *P, int Width)
{
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
  
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  float Pvalue = 0;
  for (int ph = 0; ph < ceil(Width/(float)TILE_WIDTH); ++ph) {
    if ((Row < Width) && (ph * TILE_WIDTH + tx) < Width)
      Mds[ty][tx] = M[Row * Width + ph * TILE_WIDTH + tx];
    
    if ((ph * TILE_WIDTH + ty) < Width && Col < Width)
      Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];

    __syncthreads();
  
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
   }
   __syncthreads();
  }

  if ((Row < Width) && (Col < Width))
    P[Row * Width + Col] = Pvalue;
}

__global__
void MatrixMulKernel(float *M, float *N, float *P, int Width)
{
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;

  if ((Row < Width) && (Col < Width)) {
    float Pvalue = 0;
    for (int k = 0; k < Width; ++k) {
      Pvalue += M[Row * Width + k] * N[k * Width + Col];
    }
    P[Row * Width + Col] = Pvalue; 
  }
}

int main(void)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int N = 4 * 4; // 2x2 matrix
  int size = sizeof(float) * N;
  float *h_M = (float *)malloc(size);
  float *h_N = (float *)malloc(size);

  for (int i = 0; i < N; ++i) {
    h_M[i] = (float)i;
    h_N[i] = (float)i; 
  }

  float *h_P = (float *)malloc(size);

  float *d_M, *d_N, *d_P;
  hipMalloc((void **)&d_M, size);
  hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_N, size);
  hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_P, size);

  dim3 dimGrid(2, 2, 1);
  dim3 dimBlock(2, 2, 1);

  printf("Tiled:\n");

  hipEventRecord(start);
  TiledMatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, 4);  
  hipEventRecord(stop);

  hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  for (int i = 0; i < N; ++i)
    printf("%f\n", h_P[i]);

  printf("Time: %f\n", milliseconds);


  free(h_M);
  free(h_N);
  free(h_P);
}
